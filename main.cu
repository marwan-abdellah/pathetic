#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <limits>
#include <conio.h>

/* TODO:
 * Add comments
 * use __shared__ for blocks
 * split into modules (figure out the structure)
 * port camera->world code
 * clipping the image plane
 * write a readme
 */

struct Sphere {
	float3 position;
	float radius;
	float3 colour;
	float3 emission;

	Sphere() {
		position = make_float3(0.0f, 0.0f, 0.0f);
		colour = make_float3(0.0f, 0.0f, 0.0f);
		emission = make_float3(0.0f, 0.0f, 0.0f);
		radius = 0.0f;
	}
};

struct Ray {
	float3 origin;
	float3 direction;
};

struct Scene {
	int size;
	Sphere* spheres;
};

struct Bitmap {
	int width;
	int height;
	int stride;
	float3 *elements;
};

float3 getElement(const Bitmap b, int row, int col) {
	return b.elements[row * b.stride + col];
}

__device__ void setElement(Bitmap b, int row, int col, float3 value) {
	b.elements[row * b.stride + col] = value;
}

__device__ bool intersect(Ray r, Sphere s, float &distance) {
	float dx = r.origin.x - s.position.x;
	float dy = r.origin.y - s.position.y;
	float dz = r.origin.z - s.position.z;

	float b = 2.0f * (
		  r.direction.x * dx
		+ r.direction.y * dy
		+ r.direction.z * dz);
	float c = dx * dx + dy * dy + dz * dz - s.radius * s.radius;
	float d = b * b - 4.0f * c;

	if (d < 0.0f) return false;

	float sqrtd = sqrtf(d);

	distance = (-b - sqrtd) * 0.5f;
	if (distance > 0.0f) return true;

	distance = (-b + sqrtd) * 0.5f;
	if (distance > 0.0f) return true;
	return false;
}

__device__ int sceneIntersect(Ray r, Scene scene, float &closest_dist) {
	int closest_id = -1;
	closest_dist = FLT_MAX;

	for (int i = 0; i < scene.size; i++) {
		float dist;
		if (!intersect(r, scene.spheres[i], dist)) continue;

		if (dist < closest_dist) {
			closest_dist = dist;
			closest_id = i;
		}
	}

	return closest_id;
}


__device__ inline float3 normalize(float3 v) {
	float invdist = rsqrtf(v.x * v.x + v.y * v.y + v.z * v.z);

	float3 result = {
		v.x * invdist,
		v.y * invdist,
		v.z * invdist
	};

	return result;
}

__device__ float3 hemisphereSample(float3 normal, float &dot, hiprandState_t *state) {
	float3 result;
	do {
		result.x = hiprand_uniform(state) * 2.0f - 1.0f;
		result.y = hiprand_uniform(state) * 2.0f - 1.0f;
		result.z = hiprand_uniform(state) * 2.0f - 1.0f;

		dot = result.x * normal.x
		+ result.y * normal.y
		+ result.z * normal.z;
	} while (dot > 1.0f || dot <= 0.0f);

	return normalize(result);
}

__device__ float3 traceRay(Ray ray, Scene scene, int level, hiprandState_t *state) {
	float3 result = {0.0f, 0.0f, 0.0f};
	float3 factor = {1.0f, 1.0f, 1.0f};

	for (int i = 0; i <= level; i++) {
		float dist;
		int id = sceneIntersect(ray, scene, dist);

		if (id == -1) break;
		Sphere sphere = scene.spheres[id];

		ray.origin = make_float3(
			ray.origin.x + ray.direction.x * dist,
			ray.origin.y + ray.direction.y * dist,
			ray.origin.z + ray.direction.z * dist);

		float3 normal = {
				ray.origin.x - sphere.position.x,
				ray.origin.y - sphere.position.y,
				ray.origin.z - sphere.position.z
		};

		normal = normalize(normal);

		float dot;
		ray.direction = hemisphereSample(normal, dot, state);
		
		ray.origin.x += ray.direction.x * 0.001;
		ray.origin.y += ray.direction.y * 0.001;
		ray.origin.z += ray.direction.z * 0.001;

		result.x += factor.x * sphere.emission.x;
		result.y += factor.y * sphere.emission.y;
		result.z += factor.z * sphere.emission.z;

		factor.x *= sphere.colour.x * dot;
		factor.y *= sphere.colour.y * dot;
		factor.z *= sphere.colour.z * dot;
	}
	
	return result;
}

__global__ void setupPRNGs(hiprandState *state) {
	int id = threadIdx.x + blockDim.x * threadIdx.y;
	hiprand_init(1234, id, 0, &state[id]);
}

__global__ void rayTrace(Scene scene, Bitmap bitmap, float3 cameraPos, float3 imagePlaneCentre, float3 xPixel, float3 yPixel, int noSamples, hiprandState *states) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int id = threadIdx.x + blockDim.x * threadIdx.y;

	hiprandState state = states[id];

	if (x >= bitmap.width || y >= bitmap.height) return;

	float xFactor = x - 0.5f * bitmap.width + 0.5f;
	float yFactor = y - 0.5f * bitmap.height + 0.5f;
	
	float3 rayDir = {
		imagePlaneCentre.x + xPixel.x * xFactor + yPixel.x * yFactor,
		imagePlaneCentre.y + xPixel.y * xFactor + yPixel.y * yFactor,
		imagePlaneCentre.z + xPixel.z * xFactor + yPixel.z * yFactor,
	};

	rayDir = normalize(rayDir);

	Ray ray;
	ray.origin = cameraPos;
	ray.direction = rayDir;

	float3 result = {0.0f, 0.0f, 0.0f};

	for (int i = 0; i < noSamples; i++) {
		float3 sample = traceRay(ray, scene, 5, &state);
		result.x += sample.x;
		result.y += sample.y;
		result.z += sample.z;
	}

	float factor = 1.0f/noSamples;

	result.x *= factor;
	result.y *= factor;
	result.z *= factor;

	setElement(bitmap, y, x, result);
}

int clamp(float component) {
	component = component / (1.0f + component);
	int c = (int)roundf(component * 255.0f);
	if (c > 255) c = 255;
	if (c < 0) c = 0;

	return c;
}

void saveBitmapToFile(Bitmap bitmap, char *filename) {
	//Bitmap saving code inspired by
	//http://stackoverflow.com/questions/2654480/writing-bmp-image-in-pure-c-c-without-other-libraries

	int filesize = 3 * bitmap.width * bitmap.height;
	unsigned char bfheader [14] = {'B','M',0,0,0,0,0,0,0,0,54,0,0,0};
	unsigned char biheader [40] = {40,0,0,0,0,0,0,0,0,0,0,0,1,0,24,0};
    
	bfheader[ 2] = (unsigned char)(filesize    );
	bfheader[ 3] = (unsigned char)(filesize>> 8);
	bfheader[ 4] = (unsigned char)(filesize>>16);
	bfheader[ 5] = (unsigned char)(filesize>>24);

	biheader[ 4] = (unsigned char)(bitmap.width);
	biheader[ 5] = (unsigned char)(bitmap.width>> 8);
	biheader[ 6] = (unsigned char)(bitmap.width>>16);
	biheader[ 7] = (unsigned char)(bitmap.width>>24);
	biheader[ 8] = (unsigned char)(bitmap.height    );
	biheader[ 9] = (unsigned char)(bitmap.height>> 8);
	biheader[10] = (unsigned char)(bitmap.height>>16);
	biheader[11] = (unsigned char)(bitmap.height>>24);
    
	//Open the output file and write the header
	FILE* output;

	output = fopen(filename, "wb");

	fwrite(&bfheader, 1, sizeof(bfheader), output);
	fwrite(&biheader, 1, sizeof(biheader), output);

	//Output the bitmap

	//BMP requires every row to be padded to 4 bytes
	int padding = 4 - ((bitmap.width * 3) % 4);
	if (padding == 4) padding = 0; //side effect if width mod 4 = 0 :)

	for (int i = bitmap.height - 1; i >= 0; i--) {
		for (int j = 0; j < bitmap.width; j++) {
			//Write the B, G, R to the output
			unsigned char clamped;
			clamped = clamp(getElement(bitmap, i, j).z);
			fwrite(&clamped, 1, 1, output);

			clamped = clamp(getElement(bitmap, i, j).y);
			fwrite(&clamped, 1, 1, output);

			clamped = clamp(getElement(bitmap, i, j).x);
			fwrite(&clamped, 1, 1, output);
		}
        
		//Pad the row to 4 bytes
		for (int p = 0; p < padding; p++) fputc(0, output);
	}

	fclose(output);
}

int main() {
	Scene testScene;
	testScene.size = 2;
	testScene.spheres = new Sphere[2];
	testScene.spheres[0].position.x = -5.0f;
	testScene.spheres[0].position.y = 0.0f;
	testScene.spheres[0].position.z = 10.0f;
	testScene.spheres[0].radius = 5.0f;
	testScene.spheres[0].colour.x = 1.0f;
	testScene.spheres[0].colour.y = 0.0f;
	testScene.spheres[0].colour.z = 0.0f;
	testScene.spheres[0].emission.x = 1.0f;
	testScene.spheres[0].emission.y = 0.5f;
	testScene.spheres[0].emission.z = 0.5f;
	testScene.spheres[1].position.x = 5.0f;
	testScene.spheres[1].position.y = 0.0f;
	testScene.spheres[1].position.z = 10.0f;
	testScene.spheres[1].radius = 5.0f;
	testScene.spheres[1].colour.x = 1.0f;
	testScene.spheres[1].colour.y = 1.0f;
	testScene.spheres[1].colour.z = 1.0f;

	Scene deviceScene = testScene;

	int resX = 640;
	int resY = 480;

	Bitmap deviceBitmap;
	deviceBitmap.height = resY;
	deviceBitmap.width = resX;
	deviceBitmap.stride = resX;

	float3 cameraPos = {0.0f, 0.0f, -10.0f};
	float3 cameraToImagePlane = {0.0f, 0.0f, 10.0f};
	float3 xPixel = {16.0f/resX, 0.0f, 0.0f};
	float3 yPixel = {0.0f, -12.0f/resY, 0.0f};

	dim3 threadsPerBlock(16, 16);

	dim3 numBlocks(resX / threadsPerBlock.x + (resX % threadsPerBlock.x == 0? 0 : 1),
		resY / threadsPerBlock.y + (resY % threadsPerBlock.y == 0? 0 : 1));

	hiprandState *prngStates;

	hipError_t err;

	printf("Allocating the array on the CUDA device...\n");
	hipMalloc(&(deviceScene.spheres), testScene.size * sizeof(Sphere));
	hipMalloc(&(deviceBitmap.elements), resX * resY * sizeof(float3));
	hipMalloc(&prngStates, sizeof(hiprandState) * threadsPerBlock.x * threadsPerBlock.y);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while allocating memory on the GPU!\n");
		return -1;
	}

	printf("Initialising the random number generators...\n");
	setupPRNGs<<<numBlocks, threadsPerBlock>>>(prngStates);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while seeding the PRNGs!\n");
		return -1;
	}


	printf("Copying the scene to the CUDA device...\n");
	hipMemcpy(deviceScene.spheres, testScene.spheres, testScene.size * sizeof(Sphere), hipMemcpyHostToDevice);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while copying the scene!\n");
		return -1;
	}

	printf("Starting the kernels...\n");
	rayTrace<<<numBlocks, threadsPerBlock>>>(deviceScene, deviceBitmap, cameraPos, cameraToImagePlane, xPixel, yPixel, 100, prngStates);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while executing the child kernel!\n");
		return -1;
	}

	//...and now we wait.
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while executing the child kernel!\n");
		return -1;
	}

	printf("Job finished, copying the buffer back...\n");

	Bitmap localBitmap = deviceBitmap;
	localBitmap.elements = new float3[resX * resY];

	hipMemcpy(localBitmap.elements, deviceBitmap.elements, sizeof(float3) * resX * resY, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		printf("Error while retrieving the bitmap!");
		return -1;
	}

	printf("Tonemapping and saving...\n");

	saveBitmapToFile(localBitmap, "test.bmp");

	printf("Saving complete.\n");
	hipDeviceReset();
}