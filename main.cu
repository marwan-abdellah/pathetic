#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <limits>
#include <conio.h>

/* TODO:
 * use __shared__ for blocks
 * split into modules (figure out the structure)
 * port camera->world code
 * clipping the image plane
 * write a readme
 */

struct Sphere {
	float3 position;
	float radius;
	float3 colour;
	float3 emission;

	Sphere() {
		position = make_float3(0.0f, 0.0f, 0.0f);
		colour = make_float3(0.0f, 0.0f, 0.0f);
		emission = make_float3(0.0f, 0.0f, 0.0f);
		radius = 0.0f;
	}

	Sphere(float3 p, float r, float3 c, float3 e) {
		position = p;
		radius = r;
		colour = c;
		emission = e;
	}
};

//A scene so far is just an array of spheres.
struct Scene {
	int size;
	Sphere* spheres;
};

//Represents an unnormalized bitmap.
struct Bitmap {
	int width;
	int height;
	int stride;
	float3 *elements;
};

__device__ inline float3 getElement(const Bitmap b, int row, int col) {
	return b.elements[row * b.stride + col];
}

float3 hostGetElement(const Bitmap b, int row, int col) {
	return b.elements[row * b.stride + col];
}

__device__ inline void setElement(Bitmap b, int row, int col, float3 value) {
	b.elements[row * b.stride + col] = value;
}

//Returns true and sets the distance if the ray this thread is responsible for 
//intersects this sphere, false otherwise (and the distance is undefined then).
__device__ bool intersect(int threadId, Sphere s, float &distance) {
	//The ray's position and direction are stored in rayBuffer[threadId * 2] 
	//and rayBuffer[threadId * 2 + 1], respectively.
	extern __shared__ float3 rayBuffer[];

	//Find the discriminant of the quadratic equation (standard intersection test)
	float dx = rayBuffer[threadId * 2].x - s.position.x;
	float dy = rayBuffer[threadId * 2].y - s.position.y;
	float dz = rayBuffer[threadId * 2].z - s.position.z;

	float b = 2.0f * (
		  rayBuffer[threadId * 2 + 1].x * dx
		+ rayBuffer[threadId * 2 + 1].y * dy
		+ rayBuffer[threadId * 2 + 1].z * dz);

	float d = b * b - 4.0f * (dx * dx + dy * dy + dz * dz - s.radius * s.radius);

	if (d < 0.0f) return false; //The ray misses the sphere

	//Otherwise, try the closer distance...
	float sqrtd = sqrtf(d);

	//If it's behind the camera (not the image plane for now), try the further distance...
	distance = (-b - sqrtd) * 0.5f;
	if (distance > 0.0f) return true;

	//If even this distance is behind the camera, the ray misses the sphere.
	distance = (-b + sqrtd) * 0.5f;
	if (distance > 0.0f) return true;
	return false;
}

//Goes through the scene and finds the closest sphere that intersects the ray
//this thread is responsible for. If it's found, returns its id and sets the
//closest distance variable, if not, returns -1.
__device__ int sceneIntersect(int threadId, Scene scene, float &closest_dist) {
	int closest_id = -1;
	closest_dist = FLT_MAX;

	for (int i = 0; i < scene.size; i++) {
		float dist;
		if (!intersect(threadId, scene.spheres[i], dist)) continue;

		if (dist < closest_dist) {
			closest_dist = dist;
			closest_id = i;
		}
	}

	return closest_id;
}

__device__ inline float3 normalize(float3 v) {
	float invdist = rsqrtf(v.x * v.x + v.y * v.y + v.z * v.z);

	float3 result = {
		v.x * invdist,
		v.y * invdist,
		v.z * invdist
	};

	return result;
}

//Returns a vector in the hemisphere around the normal and sets dot to the dot product
//of the result with the normal (using rejection sampling).
__device__ float3 hemisphereSample(float3 normal, float &dot, hiprandState_t *state) {
	float3 result;
	do {
		result.x = hiprand_uniform(state) * 2.0f - 1.0f;
		result.y = hiprand_uniform(state) * 2.0f - 1.0f;
		result.z = hiprand_uniform(state) * 2.0f - 1.0f;

		dot = result.x * normal.x
		+ result.y * normal.y
		+ result.z * normal.z;
	} while (dot > 1.0f || dot <= 0.0f);

	return normalize(result);
}

//Does the pathtracing of a ray recorded in the shared memory, terminates after <level> bounces.
__device__ float3 traceRay(int threadId, Scene scene, int level, hiprandState_t *state) {
	extern __shared__ float3 rayBuffer[];

	float3 result = {0.0f, 0.0f, 0.0f};	//Current accumulated pixel colour
	float3 factor = {1.0f, 1.0f, 1.0f}; //Accumulated colours of the oblects
	//this ray has hit (product), every contribution is multiplied by this
	//factor before being added to result.

#pragma unroll 128
	for (int i = 0; i <= level; i++) {
		//If nothing has been hit, terminate the procedure.
		//TODO: don't trace the initial ray (it's done at every sampling otherwise),
		//pass to the routine the actual point of intersection.
		float dist;
		int id = sceneIntersect(threadId, scene, dist);

		if (id == -1) break;
		Sphere sphere = scene.spheres[id];

		//Set the ray's position to the position of the collision with the sphere
		//(position + direction * distance).
		rayBuffer[threadId * 2] = make_float3(
			rayBuffer[threadId * 2].x + rayBuffer[threadId * 2 + 1].x * dist,
			rayBuffer[threadId * 2].y + rayBuffer[threadId * 2 + 1].y * dist,
			rayBuffer[threadId * 2].z + rayBuffer[threadId * 2 + 1].z * dist);

		//Find the normal to the collision point and normalize it.
		float3 normal = {
				rayBuffer[threadId * 2].x - sphere.position.x,
				rayBuffer[threadId * 2].y - sphere.position.y,
				rayBuffer[threadId * 2].z - sphere.position.z
		};

		normal = normalize(normal);

		//Take a random direction in the hemisphere around this normal.
		float dot;
		rayBuffer[threadId * 2 + 1] = hemisphereSample(normal, dot, state);
		
		//Perform the "epsilon shift" so that the ray doesn't collide with itself.
		rayBuffer[threadId * 2].x += rayBuffer[threadId * 2 + 1].x * 0.001;
		rayBuffer[threadId * 2].y += rayBuffer[threadId * 2 + 1].y * 0.001;
		rayBuffer[threadId * 2].z += rayBuffer[threadId * 2 + 1].z * 0.001;

		//Add the emitted light strength to the result.
		result.x += factor.x * sphere.emission.x;
		result.y += factor.y * sphere.emission.y;
		result.z += factor.z * sphere.emission.z;

		//The next contributions will be multiplied by the cosine of the diffusion angle
		//and the colour of the sphere.
		factor.x *= sphere.colour.x * dot;
		factor.y *= sphere.colour.y * dot;
		factor.z *= sphere.colour.z * dot;
	}
	
	return result;
}

//Leaves pseudorandom generator states in the device memory.
//TODO: bake randoms?
__global__ void setupPRNGs(hiprandState *state) {
	int id = threadIdx.x + blockDim.x * threadIdx.y;
	hiprand_init(1234, id, 0, &state[id]);
}

//The main raytracing kernel. Generates the initial ray and passes it to the pathtracer.
__global__ void rayTrace(Scene scene, Bitmap bitmap, float3 cameraPos, float3 imagePlaneCentre, float3 xPixel, float3 yPixel, hiprandState *states) {
	extern __shared__ float3 rayBuffer[];

	//Which pixel are we responsible for?
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//Which thread are we?
	int id = threadIdx.x + blockDim.x * threadIdx.y;

	//If we are out of the bounds of the image, stop (when eg. the block size
	//doesn't divide the image size).
	if (x >= bitmap.width || y >= bitmap.height) return;

	//Shift the coordinates because we keep track of the centre of
	//the image plane, not its corner (and we're sampling the centres
	//of the pixels).
	//TODO: maybe jitter the position of the sampling point every sample?
	float xFactor = x - 0.5f * bitmap.width + 0.5f;
	float yFactor = y - 0.5f * bitmap.height + 0.5f;
	
	//The rays are stored in a shared memory space, visible to all threads in the block.
	//The origin is stored at id * 2 and the direction - at id * 2 + 1. This is done because
	//the shared memory is faster than the thread local memory, and the thread needs to access
	//the ray quite often.

	//Construct the vector from the camera to the centre of the pixel on the image plane
	//and normalize it.
	rayBuffer[id * 2 + 1] = normalize(make_float3(
		imagePlaneCentre.x + xPixel.x * xFactor + yPixel.x * yFactor,
		imagePlaneCentre.y + xPixel.y * xFactor + yPixel.y * yFactor,
		imagePlaneCentre.z + xPixel.z * xFactor + yPixel.z * yFactor));

	//TODO: can we optimize this out?
	rayBuffer[id * 2] = cameraPos;

	//Take one illumination sample from the scene and add it to the bitmap.
	float3 current = getElement(bitmap, y, x);
	float3 sample = traceRay(id, scene, 5, &states[id]);
	current.x += sample.x;
	current.y += sample.y;
	current.z += sample.z;

	setElement(bitmap, y, x, current);
}

//Multiplies all values in the bitmap by a certain factor (used for normalizing
//after the sampling, multiplied by 1/noSamples).
__global__ void scale(Bitmap bitmap, float factor) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= bitmap.width || y >= bitmap.height) return;

	float3 pixel = getElement(bitmap, y, x);
	pixel.x *= factor;
	pixel.y *= factor;
	pixel.z *= factor;

	setElement(bitmap, y, x, pixel);
}

//Initializes the bitmap with zeroes.
__global__ void zeroBitmap(Bitmap bitmap) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= bitmap.width || y >= bitmap.height) return;

	float3 pixel = {0.0f, 0.0f, 0.0f};

	setElement(bitmap, y, x, pixel);
}

//Tonemaps a component and converts it to 0..255.
int clamp(float component) {
	component = component / (1.0f + component);
	int c = (int)roundf(component * 255.0f);
	if (c > 255) c = 255;
	if (c < 0) c = 0;

	return c;
}

//Saves a bitmap to a BMP file.
void saveBitmapToFile(Bitmap bitmap, char *filename) {
	//Bitmap saving code inspired by
	//http://stackoverflow.com/questions/2654480/writing-bmp-image-in-pure-c-c-without-other-libraries

	int filesize = 3 * bitmap.width * bitmap.height;
	unsigned char bfheader [14] = {'B','M',0,0,0,0,0,0,0,0,54,0,0,0};
	unsigned char biheader [40] = {40,0,0,0,0,0,0,0,0,0,0,0,1,0,24,0};
    
	bfheader[ 2] = (unsigned char)(filesize    );
	bfheader[ 3] = (unsigned char)(filesize>> 8);
	bfheader[ 4] = (unsigned char)(filesize>>16);
	bfheader[ 5] = (unsigned char)(filesize>>24);

	biheader[ 4] = (unsigned char)(bitmap.width);
	biheader[ 5] = (unsigned char)(bitmap.width>> 8);
	biheader[ 6] = (unsigned char)(bitmap.width>>16);
	biheader[ 7] = (unsigned char)(bitmap.width>>24);
	biheader[ 8] = (unsigned char)(bitmap.height    );
	biheader[ 9] = (unsigned char)(bitmap.height>> 8);
	biheader[10] = (unsigned char)(bitmap.height>>16);
	biheader[11] = (unsigned char)(bitmap.height>>24);
    
	//Open the output file and write the header
	FILE* output;

	output = fopen(filename, "wb");

	fwrite(&bfheader, 1, sizeof(bfheader), output);
	fwrite(&biheader, 1, sizeof(biheader), output);

	//Output the bitmap

	//BMP requires every row to be padded to 4 bytes
	int padding = 4 - ((bitmap.width * 3) % 4);
	if (padding == 4) padding = 0; //side effect if width mod 4 = 0 :)

	for (int i = bitmap.height - 1; i >= 0; i--) {
		for (int j = 0; j < bitmap.width; j++) {
			//Write the B, G, R to the output
			unsigned char clamped;
			clamped = clamp(hostGetElement(bitmap, i, j).z);
			fwrite(&clamped, 1, 1, output);

			clamped = clamp(hostGetElement(bitmap, i, j).y);
			fwrite(&clamped, 1, 1, output);

			clamped = clamp(hostGetElement(bitmap, i, j).x);
			fwrite(&clamped, 1, 1, output);
		}
        
		//Pad the row to 4 bytes
		for (int p = 0; p < padding; p++) fputc(0, output);
	}

	fclose(output);
}

int main() {
	Scene testScene;
	testScene.size = 4;
	testScene.spheres = new Sphere[testScene.size];
	
	//Simulate planes with very large spheres.
	testScene.spheres[0] = Sphere(make_float3(0, 1000, 0), 1000, make_float3(0.5, 1.0, 0.5), make_float3(0, 0, 0));
	testScene.spheres[1] = Sphere(make_float3(0, 0, 1020), 1000, make_float3(1, 1, 1), make_float3(0, 0, 0));
	testScene.spheres[2] = Sphere(make_float3(0, -5, 10), 5, make_float3(0.5, 0.5, 0.9), make_float3(0.0, 0.0, 0.0));
	testScene.spheres[3] = Sphere(make_float3(-5, -20, 20), 5, make_float3(0.5, 0.5, 0.5), make_float3(10.0, 10.0, 7.0));

	Scene deviceScene = testScene;

	int resX = 640;
	int resY = 480;

	Bitmap deviceBitmap;
	deviceBitmap.height = resY;
	deviceBitmap.width = resX;
	deviceBitmap.stride = resX;

	float3 cameraPos = {0.0f, -5.0f, -10.0f};
	float3 cameraToImagePlane = {0.0f, 0.0f, 10.0f};
	float3 xPixel = {16.0f/resX, 0.0f, 0.0f};
	float3 yPixel = {0.0f, 12.0f/resY, 0.0f};

	dim3 threadsPerBlock(16, 16);

	dim3 numBlocks(resX / threadsPerBlock.x + (resX % threadsPerBlock.x == 0? 0 : 1),
		resY / threadsPerBlock.y + (resY % threadsPerBlock.y == 0? 0 : 1));

	hiprandState *prngStates;

	hipError_t err;

	//Allocate the memory for the scene, the bitmap, the random generator state
	//and initialize the bitmap.
	printf("Allocating the array on the CUDA device...\n");
	hipMalloc(&(deviceScene.spheres), testScene.size * sizeof(Sphere));
	hipMalloc(&(deviceBitmap.elements), resX * resY * sizeof(float3));
	zeroBitmap<<<numBlocks, threadsPerBlock>>>(deviceBitmap);
	hipMalloc(&prngStates, sizeof(hiprandState) * threadsPerBlock.x * threadsPerBlock.y);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while allocating memory on the GPU!\n");
		return -1;
	}

	//Setup the random generator states for each thread in a block.
	printf("Initialising the random number generators...\n");
	setupPRNGs<<<numBlocks, threadsPerBlock>>>(prngStates);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while seeding the PRNGs!\n");
		return -1;
	}

	//Send the scene to the card memory.
	printf("Copying the scene to the CUDA device...\n");
	hipMemcpy(deviceScene.spheres, testScene.spheres, testScene.size * sizeof(Sphere), hipMemcpyHostToDevice);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error while copying the scene!\n");
		return -1;
	}
	
	//Setup the timing for the actual pathtracing.
	float time;
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int noSamples = 10000;

	//Perform noSamples samples and accumulate them in the bitmap.
	printf("Starting the kernels...\n");
	for (int i = 0; i < noSamples; i++) {
		rayTrace<<<numBlocks, threadsPerBlock, threadsPerBlock.x * threadsPerBlock.y * sizeof(float3) * 2>>>(deviceScene, deviceBitmap, cameraPos, cameraToImagePlane, xPixel, yPixel, prngStates);
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Error while executing the child kernel!\n");
			return -1;
		}

		//...and now we wait.
		hipDeviceSynchronize();
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Error while executing the child kernel!\n");
			return -1;
		}
	}

	//Stop the CUDA timer.
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//Divide each pixel in the bitmap by the number of samples.
	float factor = 1.0f/noSamples;
	scale<<<numBlocks, threadsPerBlock>>>(deviceBitmap, factor);

	printf("Job finished, copying the buffer back...\n");

	//The local bitmap is the same as the device bitmap, except that we need to actually
	//get the floating point values from the device.
	Bitmap localBitmap = deviceBitmap;
	localBitmap.elements = new float3[resX * resY];

	hipMemcpy(localBitmap.elements, deviceBitmap.elements, sizeof(float3) * resX * resY, hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess) {
		printf("Error while retrieving the bitmap!");
		return -1;
	}

	printf("Tonemapping and saving...\n");

	saveBitmapToFile(localBitmap, "test.bmp");

	printf("Saving complete.\n");
	hipDeviceReset();

	unsigned long raysTraced = (long)noSamples * (long)resX * (long)resY * 5;
	printf("Traced %lu rays in %3.1f ms, average speed: %3.1f MRays/s", raysTraced, time, (raysTraced / 1000.0f / time));

	getch();
}